#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#include <string.h>
// #include <omp.h> // Để sử dụng OpenMP Khoong dung duoc trong CUDA
#define U64 uint64_t
#define U32 uint32_t
#define U16 uint16_t
#define U8 uint8_t
#define I64 int64_t
#define I32 int32_t
#define I16 int16_t
#define I8 int8_t

#define loop(i, a, b) for (I64 i = a; i < b; i++)
#define glo_n 34
#define MAX_NUM_THREADS 1024
// const U32 glo_length = 1 << glo_n;
#define glo_length 17179869184ULL // 12 4096ULL // 16 65536ULL// 20 - 1048576 //24 16777216ULL // 30 1073741824ULL  //32 4294967296ULL //34 17179869184ULL // 36 68719476736ULL

// ------------------------ Begin kerner Functions -----------------------------
__global__ void addKernel(int *c, const int *a, const int *b, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        c[i] = a[i] + b[i];
    }
}
__global__ void supportF2(I8 *F, const I64 trust_length_block)
{
    U64 i = blockIdx.x * blockDim.x + threadIdx.x;
    F[i + trust_length_block] = F[i] ^ 1;
    // printf("i = %u dim = %u ||",i, blockDim.x);
}
/*
__global__ void supportZhegalkinGPU(U8* BLOCK,const U64 half_length_block, U8* used)
{

    U64 i = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("%u %u %u\n", blockIdx.x, blockDim.x, threadIdx.x);
    U64 idx3 = threadIdx.x;
    used[i] ^= 1;
    if (idx3 < half_length_block) {
        BLOCK[i+ half_length_block] ^= BLOCK[i];
    }
}*/
// zhegalkin
__global__ void supportZhegalkinGPU2(U8 *BLOCK, const U64 half_length_block)
{
    U64 i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i % (half_length_block << 1) < half_length_block)
        BLOCK[i + half_length_block] ^= BLOCK[i];
}
// real
__global__ void supportRealGPU(I8 *BLOCK, const U64 half_length_block)
{
    U64 i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i % (half_length_block << 1) < half_length_block)
        BLOCK[i + half_length_block] -= BLOCK[i];
}
__global__ void supportFourierGPU(I8 *BLOCK, const U64 half_length_block)
{
    U64 i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i % (half_length_block << 1) < half_length_block)
    {
        BLOCK[i] += BLOCK[i + half_length_block];
        BLOCK[i + half_length_block] = BLOCK[i] - 2 * BLOCK[i + half_length_block];
    }
}
// ------------------------ End kerner Functions -----------------------------

// ======================== Begin discret function ===========================
void fastFindCoefficientsZhegalkinGPU(U8 *lst_coefficients)
{
    U64 half_len;
    U64 full_len;
    U64 num_block;
    U8 flag;
    flag = glo_length <= MAX_NUM_THREADS;
    if (flag)
    {
        loop(idx1, 0, glo_n)
        {
            // cudaMemset(global_table_GPU, 1, glo_length * sizeof(U8));
            half_len = 1ULL << idx1;
            full_len = half_len << 1;
            num_block = glo_length / full_len;
            supportZhegalkinGPU2<<<num_block, full_len>>>(lst_coefficients, half_len);
            // Wait for GPU to finish before accessing on host
            hipDeviceSynchronize();
        }
    }
    else
    {
        loop(idx1, 0, 10)
        {
            // cudaMemset(global_table_GPU, 1, glo_length * sizeof(U8));
            half_len = 1ULL << idx1;
            full_len = half_len << 1;
            num_block = glo_length / full_len;
            supportZhegalkinGPU2<<<num_block, full_len>>>(lst_coefficients, half_len);
            // Wait for GPU to finish before accessing on host
            hipDeviceSynchronize();
        }
        loop(idx1, 10, glo_n)
        {
            // cudaMemset(global_table_GPU, 1, glo_length * sizeof(U8));
            half_len = 1ULL << idx1;
            // full_len = MAX_NUM_THREADS;
            num_block = glo_length / MAX_NUM_THREADS;
            supportZhegalkinGPU2<<<num_block, MAX_NUM_THREADS>>>(lst_coefficients, half_len);
            // Wait for GPU to finish before accessing on host
            hipDeviceSynchronize();
        }
    }
}

// Real
void fastFindCoefficientsRealGPU(I8 *lst_coefficients)
{
    U64 half_len;
    U64 full_len;
    U64 num_block;
    U8 flag;
    flag = glo_length <= MAX_NUM_THREADS;
    if (flag)
    {
        loop(idx1, 0, glo_n)
        {
            // cudaMemset(global_table_GPU, 1, glo_length * sizeof(U8));
            half_len = 1ULL << idx1;
            full_len = half_len << 1;
            num_block = glo_length / full_len;
            supportRealGPU<<<num_block, full_len>>>(lst_coefficients, half_len);
            // Wait for GPU to finish before accessing on host
            hipDeviceSynchronize();
        }
    }
    else
    {
        loop(idx1, 0, 10)
        {
            // cudaMemset(global_table_GPU, 1, glo_length * sizeof(U8));
            half_len = 1ULL << idx1;
            full_len = half_len << 1;
            num_block = glo_length / full_len;
            supportRealGPU<<<num_block, full_len>>>(lst_coefficients, half_len);
            // Wait for GPU to finish before accessing on host
            hipDeviceSynchronize();
        }
        loop(idx1, 10, glo_n)
        {
            // cudaMemset(global_table_GPU, 1, glo_length * sizeof(U8));
            half_len = 1ULL << idx1;
            // full_len = MAX_NUM_THREADS;
            num_block = glo_length / MAX_NUM_THREADS;
            supportRealGPU<<<num_block, MAX_NUM_THREADS>>>(lst_coefficients, half_len);
            // Wait for GPU to finish before accessing on host
            hipDeviceSynchronize();
        }
    }
}
void fastFindCoefficientsFourierGPU(I8 *lst_coefficients)
{
    U64 half_len;
    U64 full_len;
    U64 num_block;
    U8 flag;
    flag = glo_length <= MAX_NUM_THREADS;
    if (flag)
    {
        loop(idx1, 0, glo_n)
        {
            // cudaMemset(global_table_GPU, 1, glo_length * sizeof(U8));
            half_len = 1ULL << idx1;
            full_len = half_len << 1;
            num_block = glo_length / full_len;
            supportFourierGPU<<<num_block, full_len>>>(lst_coefficients, half_len);
            // Wait for GPU to finish before accessing on host
            hipDeviceSynchronize();
        }
    }
    else
    {
        loop(idx1, 0, 10)
        {
            // cudaMemset(global_table_GPU, 1, glo_length * sizeof(U8));
            half_len = 1ULL << idx1;
            full_len = half_len << 1;
            num_block = glo_length / full_len;
            supportFourierGPU<<<num_block, full_len>>>(lst_coefficients, half_len);
            // Wait for GPU to finish before accessing on host
            hipDeviceSynchronize();
        }
        loop(idx1, 10, glo_n)
        {
            // cudaMemset(global_table_GPU, 1, glo_length * sizeof(U8));
            half_len = 1ULL << idx1;
            // full_len = MAX_NUM_THREADS;
            num_block = glo_length / MAX_NUM_THREADS;
            supportFourierGPU<<<num_block, MAX_NUM_THREADS>>>(lst_coefficients, half_len);
            // Wait for GPU to finish before accessing on host
            hipDeviceSynchronize();
        }
    }
}
// ======================== End discret function ===========================

// ======================== Begin functions to create F1,F2 ===========================
// phan tao ham F
I64 f = 0;
// U8 *global_table_GPU = (U8 *)calloc(glo_length, sizeof(U8));
// Gán giá trị 1 cho tất cả các phần tử trong mảng bằng hàm memset
I8 *global_table_GPU;
// U8* global_table_CPU;
bool create_dynamic_array_cuda()
{
    hipError_t cudaStatus;
    cudaStatus = hipMallocManaged(&global_table_GPU, glo_length * sizeof(I8));

    if (cudaStatus == hipSuccess)
    {
        printf("Cấp phát động thành công cho global_table.\n");
    }
    else
    {
        printf("Cấp phát động không thành công cho global_table: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }
    hipMemset(global_table_GPU, 1, glo_length * sizeof(I8));
    return 1;
}

bool create_dynamic_array_cuda_for_F2()
{
    hipError_t cudaStatus;
    cudaStatus = hipMallocManaged(&global_table_GPU, glo_length * sizeof(I8));
    if (cudaStatus == hipSuccess)
    {
        printf("Cấp phát động thành công cho global_table.\n");
    }
    else
    {
        printf("Cấp phát động không thành công cho global_table: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }
    hipMemset(global_table_GPU, 0, glo_length * sizeof(I8));
    return 1;
}
void recursive(I64 t1, I16 num_loop)
{
    if (num_loop)
    {
        I64 t2 = 1 << num_loop;
        while (t2 < t1)
        {
            f ^= t2;
            recursive(t2, num_loop - 1);
            f ^= t2;
            t2 <<= 1;
        }
    }
    else
    {
        I64 t2 = 1;
        while (t2 < t1)
        {
            f ^= t2;
            // printf("%u,", f);
            global_table_GPU[f] = 0;
            f ^= t2;
            t2 <<= 1;
            //++c;
        }
    }
}
void calcF1SpAuto()
{
    global_table_GPU[0] = 0;
    f = 0;
    loop(t, 0, (glo_n >> 1) - 1)
    {
        // c = 0;
        recursive(glo_length, t);
        // printf("t = %u ; c = %u\n", t, c);
    }
}
// tao F2
bool calcF2CPU()
{
    global_table_GPU[0] = 1;
    global_table_GPU[1] = 0;
    U64 current_leng = 2;
    while (current_leng < glo_length)
    {
        for (I64 i = 0; i < current_leng; i++)
        {
            global_table_GPU[i + current_leng] = global_table_GPU[i] ^ 1;
            // printf("i = %i,", i);
        }
        current_leng <<= 1;
    }
    return true;
    // return global_table_GPU;
}
bool calcF2GPU()
{
    global_table_GPU[0] = 1;
    global_table_GPU[1] = 0;
    U64 current_leng = 2;
    U16 default_length_block = 1 << 8;
    U32 num_block;
    while (current_leng < glo_length)
    {
        // for (I64 i = 0; i < current_leng; i++)
        //{
        //     global_table_GPU[i + current_leng] = global_table_GPU[i] ^ 1;
        // printf("i = %i,", i);
        //}
        num_block = current_leng / default_length_block + 1;
        supportF2<<<num_block, default_length_block>>>(global_table_GPU, current_leng);
        hipDeviceSynchronize();
        current_leng <<= 1;
    }
    return true;
    // return global_table_GPU;
}
// ======================== End functions to create F1,F2 ===========================

// ======================== Begin functions CODE CPU ===========================
// CODE CPU
void fastFindCoefficientsZhegalkinCPU(U8 *lst_coefficients)
{
    U64 half_len;
    U64 full_len;
    U64 num_turns;
    loop(idx1, 0, glo_n)
    {
        half_len = 1ULL << idx1;
        full_len = half_len << 1;
        num_turns = glo_length / full_len;
        loop(idx2, 0, num_turns)
        {
            loop(idx3, 0, half_len)
                lst_coefficients[idx2 * full_len + half_len + idx3] ^= lst_coefficients[idx2 * full_len + idx3];
        }
    }
}
void fastFindCoefficientsRealCPU(I8 *lst_coefficients)
{
    U64 half_len;
    U64 full_len;
    U64 num_turns;
    loop(idx1, 0, glo_n)
    {
        half_len = 1ULL << idx1;
        full_len = half_len << 1;
        num_turns = glo_length / full_len;
        loop(idx2, 0, num_turns)
        {
            loop(idx3, 0, half_len)
                lst_coefficients[idx2 * full_len + half_len + idx3] -= lst_coefficients[idx2 * full_len + idx3];
        }
    }
}
void fastFindCoefficientsFourierCPU(int8_t *lst_coefficients)
{
    U64 half_len;
    U64 full_len;
    U64 num_turns;
    loop(idx1, 0, glo_n)
    {
        half_len = 1ULL << idx1;
        full_len = half_len << 1;
        num_turns = glo_length / full_len;
        loop(idx2, 0, num_turns)
        {
            loop(idx3, 0, half_len)
            {
                lst_coefficients[idx2 * full_len + idx3] += lst_coefficients[idx2 * full_len + idx3 + half_len];
                lst_coefficients[idx2 * full_len + half_len + idx3] = lst_coefficients[idx2 * full_len + idx3] - 2 * lst_coefficients[idx2 * full_len + half_len + idx3];
            }
        }
    }
}
U8 *global_table_CPU;
void create_dynamic_arrayCPU()
{
    global_table_CPU = (U8 *)malloc(glo_length * sizeof(U8)); // Cấp phát bộ nhớ cho mảng
    // memset(global_table_GPU, 1, glo_length * sizeof(I8));
}

// ======================== End functions CODE CPU ===========================

int main()
{

    // int size = 1000;
    // U8* a;
    clock_t start, end;
    double cpu_time_used;

    // U8 b[] = { 0,1,0,1,0,0,1,1,0,1,1,0,1,0,0,1 };
    //  Allocate Unified Memory – accessible from CPU or GPU
    // cudaMallocManaged(&a, glo_length * sizeof(U8));
    // cudaMemset(a, 1, glo_length * sizeof(U8));

    // GPU
    // giai doan 1
    U8 flag = create_dynamic_array_cuda_for_F2();
    if (flag == 0)
        return -17;
    printf("start\n");

    start = clock();
    calcF2GPU();
    end = clock();
    cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    printf("Thời gian thực thi giai đoạn 1(GPU): %f giây\n", cpu_time_used);

    // cudaMallocManaged(&global_table_CPU, glo_length * sizeof(U8));
    /*create_dynamic_arrayCPU();
    loop(i, 0, glo_length)
        global_table_CPU[i] = global_table_GPU[i];


   start = clock();
   fastFindCoefficientsZhegalkinCPU(global_table_CPU);
   end = clock();
   cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
   printf("Thời gian thực thi giai đoạn 2(CPU): %f giây\n", cpu_time_used);
   printf("\n");
   */
    //  giai doan 2

    start = clock();
    fastFindCoefficientsFourierGPU(global_table_GPU);
    // printf("finish GPU\n");
    // fastFindCoefficientsZhegalkinCPU(global_table_CPU);
    end = clock();
    cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    printf("Thời gian thực thi giai đoạn 2(GPU): %f giây\n", cpu_time_used);
    printf("\n");
    /*loop(i, 0, glo_length)
        if (global_table_GPU[i] != global_table_CPU[i])
            printf("Khac nhau tai i = %d", i);

    printf("");*/
    // CPU

    // Wait for GPU to finish before accessing on host
    // cudaDeviceSynchronize();
    /*for (int i = 0; i < glo_length; i++) {
        printf("%u,", a[i]);
    }*/
    // Free memory
    hipFree(global_table_GPU);
    // cudaFree(global_table_CPU);
    // free(F2);
    return 0;
}
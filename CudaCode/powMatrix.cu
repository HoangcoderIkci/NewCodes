#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#include <string.h>
// #include <omp.h> // Để sử dụng OpenMP Khoong dung duoc trong CUDA
#define U64 uint64_t
#define U32 uint32_t
#define U16 uint16_t
#define U8 uint8_t
#define I64 int64_t
#define I32 int32_t
#define I16 int16_t
#define I8 int8_t
#define F16 float
#define loop(i, a, b) for (I64 i = a; i < b; i++)
#define glo_n 26
#define MAX_NUM_THREADS 1024
// const U32 glo_length = 1 << glo_n;
#define glo_length 676 // 12 4096ULL // 16 65536ULL// 20 - 1048576 //24 16777216ULL // 30 1073741824ULL  //32 4294967296ULL //34 17179869184ULL // 36 68719476736ULL

// ------------------------ Begin kerner Functions -----------------------------
__global__ void squareMatrix(float *matrix)
{
    // int i = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.x;
    int col = threadIdx.x;
    int length = blockDim.x;
    float newVal = 0;
    loop(i, 0, length)
    {
        newVal += matrix[row * length + i] * matrix[i * length + col];
    }
    matrix[row * length + col] = newVal;
}
F16 *global_table_GPU;
bool create_dynamic_array_cuda_for_F2()
{
    hipError_t cudaStatus;
    cudaStatus = hipMallocManaged(&global_table_GPU, glo_length * sizeof(F16));
    if (cudaStatus == hipSuccess)
    {
        printf("Cấp phát động thành công cho global_table.\n");
    }
    else
    {
        printf("Cấp phát động không thành công cho global_table: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }
    // cudaMemset(global_table_GPU, 0, glo_length * sizeof(I8));
    return 1;
}

int main()
{
    printf("start\n");
    float test[glo_n][glo_n] = {
        {0.005, 0.025, 0.035, 0.065, 0.003, 0.014, 0.035, 0.005, 0.040, 0.005, 0.008, 0.078, 0.003, 0.210, 0.003, 0.018, 0.000, 0.104, 0.095, 0.160, 0.001, 0.031, 0.010, 0.001, 0.011, 0.001},
        {0.081, 0.000, 0.000, 0.000, 0.344, 0.000, 0.000, 0.000, 0.050, 0.013, 0.000, 0.138, 0.000, 0.000, 0.069, 0.000, 0.000, 0.094, 0.025, 0.013, 0.081, 0.000, 0.000, 0.000, 0.094, 0.000},
        {0.113, 0.000, 0.025, 0.004, 0.243, 0.000, 0.000, 0.116, 0.060, 0.000, 0.035, 0.032, 0.004, 0.000, 0.176, 0.011, 0.000, 0.035, 0.000, 0.099, 0.039, 0.000, 0.000, 0.000, 0.011, 0.000},
        {0.100, 0.040, 0.022, 0.012, 0.162, 0.045, 0.007, 0.022, 0.140, 0.000, 0.002, 0.010, 0.037, 0.015, 0.040, 0.010, 0.000, 0.052, 0.045, 0.132, 0.047, 0.012, 0.037, 0.000, 0.007, 0.000},
        {0.065, 0.016, 0.043, 0.097, 0.040, 0.031, 0.015, 0.012, 0.039, 0.001, 0.003, 0.043, 0.042, 0.114, 0.027, 0.029, 0.005, 0.149, 0.116, 0.051, 0.007, 0.020, 0.024, 0.009, 0.004, 0.000},
        {0.073, 0.011, 0.019, 0.004, 0.073, 0.080, 0.004, 0.011, 0.115, 0.008, 0.000, 0.042, 0.004, 0.000, 0.195, 0.000, 0.000, 0.099, 0.031, 0.179, 0.023, 0.011, 0.011, 0.000, 0.008, 0.000},
        {0.101, 0.020, 0.015, 0.010, 0.176, 0.005, 0.015, 0.075, 0.090, 0.000, 0.000, 0.025, 0.005, 0.020, 0.106, 0.005, 0.005, 0.101, 0.045, 0.106, 0.045, 0.000, 0.025, 0.000, 0.005, 0.000},
        {0.187, 0.002, 0.006, 0.000, 0.501, 0.009, 0.002, 0.011, 0.106, 0.000, 0.000, 0.000, 0.006, 0.004, 0.082, 0.002, 0.000, 0.006, 0.019, 0.033, 0.011, 0.000, 0.009, 0.000, 0.006, 0.000},
        {0.051, 0.009, 0.066, 0.030, 0.032, 0.012, 0.014, 0.004, 0.000, 0.000, 0.003, 0.049, 0.032, 0.260, 0.072, 0.015, 0.001, 0.059, 0.102, 0.151, 0.001, 0.028, 0.000, 0.005, 0.000, 0.004},
        {0.188, 0.000, 0.000, 0.000, 0.312, 0.000, 0.000, 0.000, 0.062, 0.000, 0.000, 0.000, 0.000, 0.000, 0.250, 0.000, 0.000, 0.000, 0.000, 0.000, 0.188, 0.000, 0.000, 0.000, 0.000, 0.000},
        {0.024, 0.000, 0.000, 0.000, 0.268, 0.000, 0.000, 0.000, 0.317, 0.000, 0.000, 0.000, 0.000, 0.049, 0.049, 0.000, 0.000, 0.000, 0.146, 0.049, 0.024, 0.000, 0.049, 0.000, 0.024, 0.000},
        {0.125, 0.006, 0.014, 0.034, 0.177, 0.020, 0.014, 0.006, 0.120, 0.003, 0.003, 0.151, 0.006, 0.006, 0.071, 0.003, 0.003, 0.006, 0.046, 0.066, 0.026, 0.000, 0.003, 0.000, 0.094, 0.000},
        {0.214, 0.058, 0.004, 0.000, 0.263, 0.000, 0.000, 0.012, 0.152, 0.000, 0.000, 0.000, 0.029, 0.004, 0.070, 0.074, 0.004, 0.008, 0.049, 0.012, 0.033, 0.000, 0.004, 0.000, 0.008, 0.000},
        {0.056, 0.013, 0.063, 0.162, 0.084, 0.025, 0.141, 0.016, 0.040, 0.001, 0.008, 0.008, 0.012, 0.009, 0.072, 0.009, 0.001, 0.009, 0.059, 0.165, 0.008, 0.001, 0.020, 0.000, 0.016, 0.000},
        {0.011, 0.018, 0.021, 0.026, 0.008, 0.144, 0.005, 0.008, 0.021, 0.000, 0.000, 0.029, 0.062, 0.202, 0.020, 0.035, 0.000, 0.137, 0.035, 0.063, 0.083, 0.024, 0.042, 0.000, 0.006, 0.002},
        {0.105, 0.006, 0.000, 0.000, 0.204, 0.000, 0.000, 0.022, 0.044, 0.000, 0.000, 0.083, 0.006, 0.000, 0.149, 0.050, 0.000, 0.182, 0.077, 0.039, 0.033, 0.000, 0.000, 0.000, 0.000, 0.000},
        {0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 1.000, 0.000, 0.000, 0.000, 0.000, 0.000},
        {0.122, 0.012, 0.023, 0.034, 0.247, 0.006, 0.012, 0.012, 0.113, 0.001, 0.015, 0.007, 0.038, 0.023, 0.088, 0.006, 0.000, 0.035, 0.054, 0.081, 0.009, 0.016, 0.006, 0.000, 0.041, 0.000},
        {0.098, 0.014, 0.026, 0.014, 0.110, 0.020, 0.002, 0.071, 0.113, 0.005, 0.000, 0.011, 0.017, 0.018, 0.085, 0.026, 0.009, 0.014, 0.073, 0.175, 0.053, 0.002, 0.042, 0.000, 0.006, 0.000},
        {0.059, 0.023, 0.007, 0.001, 0.078, 0.005, 0.002, 0.340, 0.130, 0.001, 0.000, 0.014, 0.010, 0.006, 0.081, 0.007, 0.000, 0.050, 0.051, 0.058, 0.022, 0.002, 0.028, 0.000, 0.025, 0.000},
        {0.044, 0.020, 0.036, 0.024, 0.036, 0.004, 0.024, 0.000, 0.036, 0.000, 0.004, 0.077, 0.020, 0.125, 0.004, 0.060, 0.000, 0.190, 0.157, 0.125, 0.000, 0.012, 0.000, 0.000, 0.000, 0.000},
        {0.061, 0.000, 0.000, 0.000, 0.626, 0.000, 0.000, 0.000, 0.243, 0.000, 0.000, 0.000, 0.000, 0.000, 0.043, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.026, 0.000},
        {0.200, 0.006, 0.006, 0.000, 0.211, 0.000, 0.000, 0.183, 0.200, 0.000, 0.000, 0.022, 0.006, 0.044, 0.083, 0.000, 0.000, 0.000, 0.022, 0.011, 0.000, 0.000, 0.006, 0.000, 0.000, 0.000},
        {0.059, 0.000, 0.118, 0.000, 0.000, 0.059, 0.000, 0.000, 0.176, 0.000, 0.000, 0.000, 0.000, 0.000, 0.059, 0.294, 0.000, 0.000, 0.000, 0.176, 0.000, 0.000, 0.059, 0.000, 0.000, 0.000},
        {0.092, 0.033, 0.026, 0.013, 0.046, 0.079, 0.013, 0.039, 0.066, 0.000, 0.000, 0.020, 0.046, 0.033, 0.112, 0.020, 0.000, 0.026, 0.105, 0.197, 0.000, 0.000, 0.033, 0.000, 0.000, 0.000},
        {0.200, 0.000, 0.000, 0.000, 0.800, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000, 0.000}};
    create_dynamic_array_cuda_for_F2();
    loop(i, 0, glo_n)
        loop(j, 0, glo_n)
            global_table_GPU[i * glo_n + j] = test[i][j];
    int num_pow = 2;
    printf("start\n");
    loop(i, 0, num_pow)
    {
        squareMatrix<<<glo_n, glo_n>>>(global_table_GPU);
        hipDeviceSynchronize();
    }
    loop(i, 0, glo_n)
    {
        loop(j, 0, glo_n)
            printf("%f,", global_table_GPU[i * glo_n + j]);
        printf("\n");
    }
    hipFree(global_table_GPU);
    return 0;
}